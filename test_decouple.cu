#include "hip/hip_runtime.h"
#include "iostack_decouple.cuh"
#include <unordered_set>
#include <algorithm>
#include <random>
#include <assert.h>
#include <unistd.h>
#define TEST_SIZE 0x10000000
#define APP_BUF_SIZE 0x10000000
#define NUM_QUEUES_PER_SSD 128
#define NUM_SSDS 6

__device__ uint64_t **IO_buf_base;

__device__ uint64_t seed;
__global__ void gen_test_data(int ssd_id, int req_id)
{
    for (int i = 0; i < MAX_IO_SIZE / 8; i++)
    {
        seed = seed * 0x5deece66d + 0xb;
        IO_buf_base[ssd_id][i] = req_id * MAX_IO_SIZE / 8 + i;
    }
}

__global__ void check_test_data(uint64_t *app_buf, int idx)
{
    for (int i = 0; i < MAX_IO_SIZE / 8; i++)
    {
        seed = seed * 0x5deece66d + 0xb;
        if (app_buf[i] != idx * MAX_IO_SIZE / 8 + i)
        {
            printf("check failed at block %d, i = %d, read %lx, expected %x\n", idx, i, app_buf[i], idx * MAX_IO_SIZE / 8 + i);
            assert(0);
        }
    }
}

__global__ void fill_app_buf(uint64_t *app_buf)
{
    for (int i = 0; i < TEST_SIZE / 8; i++)
        app_buf[i] = 0;
}

int main()
{
    IOStack iostack(NUM_SSDS, NUM_QUEUES_PER_SSD);
    uint64_t **d_IO_buf_base = iostack.get_d_io_buf_base();
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(IO_buf_base), &d_IO_buf_base, sizeof(uint64_t **)));

    // test do_io_req
    uint64_t *app_buf;
    CHECK(hipMalloc(&app_buf, APP_BUF_SIZE));
    fill_app_buf<<<1, 1>>>(app_buf);
    int num_reqs = TEST_SIZE / MAX_IO_SIZE;
    IOReq *reqs;
    CHECK(hipMalloc(&reqs, sizeof(IOReq) * num_reqs));
    IOReq *h_reqs;
    CHECK(hipHostAlloc(&h_reqs, sizeof(IOReq) * num_reqs, hipHostMallocMapped));
    std::unordered_set<uint64_t> lbs;
    srand(time(NULL));
    uint64_t h_seed = 0;
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(seed), &h_seed, sizeof(uint64_t)));
    int percent = 1;
    clock_t clstart = clock();
    for (int i = 0; i < num_reqs; i++)
    {
        uint64_t lb;
        do
        {
            lb = ((unsigned long)rand() << 31 | rand()) % (NUM_LBS_PER_SSD * NUM_SSDS / MAX_ITEMS);
        } while (lbs.find(lb) != lbs.end());
        lbs.insert(lb);
        h_reqs[i].start_lb = lb * MAX_ITEMS;
        h_reqs[i].num_items = MAX_ITEMS;
        for (int j = 0; j < MAX_ITEMS; j++)
            // h_reqs[i].dest_addr[j] = app_buf + i * MAX_IO_SIZE / 8 + j * ITEM_SIZE / 8;
            h_reqs[i].dest_addr[j] = (app_addr_t)(app_buf + (1ll * i * MAX_IO_SIZE + j * ITEM_SIZE) % APP_BUF_SIZE / 8);
        int ssd_id = lb * MAX_ITEMS / NUM_LBS_PER_SSD;
        // printf("%d %d\n", i, ssd_id);
        // CHECK(hipMemcpyFromSymbol(&h_seed, HIP_SYMBOL(seed), sizeof(uint64_t)));
        gen_test_data<<<1, 1>>>(ssd_id, i);
        iostack.write_data(ssd_id, h_reqs[i].start_lb % NUM_LBS_PER_SSD, MAX_IO_SIZE / LB_SIZE);
        // CHECK(hipMemset(hptr[ssd_id], 0, MAX_IO_SIZE));
        // iostack.read_data(ssd_id, h_reqs[i].start_lb, MAX_IO_SIZE / LB_SIZE);
        // CHECK(hipMemcpyToSymbol(HIP_SYMBOL(seed), &h_seed, sizeof(uint64_t)));
        // check_test_data<<<1, 1>>>(hptr[ssd_id], i);
        if (i >= num_reqs / 100 * percent)
        {
            double eta = (clock() - clstart) / (double)CLOCKS_PER_SEC / percent * (100 - percent);
            fprintf(stderr, "generating test data: %d%% done, eta %.0lfs\r", percent, eta);
            percent++;
        }
    }
    CHECK(hipDeviceSynchronize());
    std::shuffle(h_reqs, h_reqs + num_reqs, std::default_random_engine(0));
    // for (int i = 0; i < 10; i++)
    //     printf("%lx\n", h_reqs[i].start_lb);
    CHECK(hipMemcpy(reqs, h_reqs, sizeof(IOReq) * num_reqs, hipMemcpyHostToDevice));

    // sleep(1);
    // uint32_t *cq = (uint32_t *)malloc(QUEUE_DEPTH * 16);
    // for (int i = 0; i < 1; i++)
    // {
    //     SSDQueuePair qp;
    //     CHECK(hipMemcpy(&qp, iostack.d_ssdqp + i, sizeof(SSDQueuePair), hipMemcpyDeviceToHost));
    //     CHECK(hipMemcpy(cq, (const void *)qp.cq, QUEUE_DEPTH * 16, hipMemcpyDeviceToHost));
    //     for (int j = 0; j < QUEUE_DEPTH; j++)
    //         if (cq[j * 4 + 3] & PHASE_MASK)
    //             printf("ssd %d warp %d cq %d phase %d\n", i / NUM_QUEUES_PER_SSD, i % NUM_QUEUES_PER_SSD, j, cq[j * 4 + 3] & PHASE_MASK);
    // }

    // printf("Press enter to start testing...\n");
    // getchar();
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start));
    fprintf(stderr, "starting do_io_req...\n");
    iostack.submit_io_req(reqs, num_reqs, 0);
    // printf("%s\n", hipGetErrorString(hipGetLastError()));
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float ms;
    CHECK(hipEventElapsedTime(&ms, start, stop));
    fprintf(stderr, "do_io_req takes %f ms\n", ms);
    fprintf(stderr, "%dB random read bandwidth: %f MiB/s\n", MAX_IO_SIZE, TEST_SIZE / (1024 * 1024) / (ms / 1000));

    // bool *h_req_processed;
    // CHECK(hipHostAlloc(&h_req_processed, sizeof(bool) * num_reqs, hipHostMallocMapped));
    // CHECK(hipMemcpy(h_req_processed, iostack.req_processed, sizeof(bool) * num_reqs, hipMemcpyDeviceToHost));
    // for (int i = 0; i < num_reqs; i++)
    //     if (!h_req_processed[i])
    //         fprintf(stderr, "req %d not processed\n", i);

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(seed), &h_seed, sizeof(uint64_t)));
    percent = 1;
    clstart = clock();
    for (int i = 0; i < num_reqs; i++)
    {
        check_test_data<<<1, 1>>>(app_buf + i * MAX_IO_SIZE / 8, i);
        if (i >= num_reqs / 100 * percent)
        {
            double eta = (clock() - clstart) / (double)CLOCKS_PER_SEC / percent * (100 - percent);
            fprintf(stderr, "checking: %d%% done, eta %.0lfs\r", percent, eta);
            percent++;
        }
    }
    CHECK(hipDeviceSynchronize());
    fprintf(stderr, "check passed!\n");
    return 0;
}